
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define T threadIdx
#define B blockIdx
#define T_W 2		//T_W => Tile Width that lowers the burden on GPU for computation

/*matrix multiplication kernels*/
// shared
__global__ void MatrixMulSh( float *Md , float *Nd , float *Pd , const int W )
{

	//These shared variables are present in shared memory that is common to all threads within a block.
	__shared__ float Mds [T_W][T_W] ;
	__shared__ float Nds [T_W][T_W] ;

	// calculate thread id
	unsigned int col = T_W*B.x + T.x;
	unsigned int row = T_W*B.y + T.y;

	//printf("---------COL OF [%d]{%d} is: %d ------- \n", B.x, T.x, col);
	//printf("---------ROW OF [%d]{%d} is: %d ------- \n", B.y, T.y, row);
	float Pvalue = 0;
	// m indicate number of phase
	for (int m = 0 ; m < W/T_W ; m++ ) {

		//printf("***** FOR M : %d ******\n", m);
		
		Mds[T.y][T.x] =  Md[row*W + (m*T_W + T.x)];

		//printf("\nMds[%d][%d] = Md [%d]\n", T.y, T.x, (row*W+(m*T_W + T.x)));

		Nds[T.y][T.x] =  Nd[ col+( m*T_W + T.y) * W ] ;

		//printf("\nNds[%d][%d] = Nd [%d]\n", T.y, T.x, (col+(m*T_W + T.y)*W));

		__syncthreads() ; //for synchronizing the threads


		for (int k = 0; k < T_W; ++k) {
			Pvalue += Mds[T.x][k] * Nds[k][T.y];
			//printf("\nPvalue += Mds[%d][%d] * Nds[%d][%d]\n", T.x, k, k, T.y);             
		}
		__syncthreads();
	}

	Pd[row*W + col] = Pvalue;
	//printf("\n~~~Pd[%d] = %d~~~\n", row*W+col, Pvalue);
}

int main () {
	const int W = 6;
	float array1_h[W][W],array2_h[W][W],M_result_array_h[W][W];
	float *array1_d,*array2_d ,*M_result_array_d ; // device array  *result_array_d
	int i , j;
	//input in host array
	//hardcoding 1 in all slots of 1st array and 2 in all slots of 2nd array
	for ( i = 0 ; i<W ; i++ ) {
		for (j = 0 ; j<W ; j++ ) {
			array1_h[i][j] = 1;
			array2_h[i][j] = 2;
		}
	}

	//create device array cudaMalloc ( (void **)&array_name, sizeofmatrixinbytes) ;

	hipMalloc((void **) &array1_d , W*W*sizeof (int) ) ;

	hipMalloc((void **) &array2_d , W*W*sizeof (int) ) ;



	//copy host array to device array; cudaMemcpy ( dest , source , W , direction )

	hipMemcpy ( array1_d , array1_h , W*W*sizeof (int) , hipMemcpyHostToDevice ) ;

	hipMemcpy ( array2_d , array2_h , W*W*sizeof (int) , hipMemcpyHostToDevice ) ;



	//allocating memory for resultent device array

	hipMalloc((void **) &M_result_array_d , W*W*sizeof (int) );

	//calling kernal

	dim3 dimBlock ( W/T_W , W/T_W ,1 ) ;

	dim3 dimThread ( T_W, T_W, 1 ) ;

#if 1

MatrixMulSh<<<dimBlock,dimThread>>> ( array1_d , array2_d ,M_result_array_d , W) ;

#endif

	// all gpu function blocked till kernel is working
	//copy back result_array_d to result_array_h

	hipMemcpy(M_result_array_h , M_result_array_d , W*W*sizeof(int),hipMemcpyDeviceToHost) ;

	hipFree(array1_d);
	hipFree(array2_d);
	hipFree(M_result_array_d);

	//printf the result array
	for ( i = 0 ; i<W ; i++ ) {
		for ( j = 0 ; j < W ; j++ ) {
			printf ("%f   ",M_result_array_h[i][j] ) ;
		}
		printf ("\n") ;
	}
	hipFree(M_result_array_h);
}
